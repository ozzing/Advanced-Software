#include "hip/hip_runtime.h"
#pragma once

#include "cuda_code.cuh"

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

#define Window 2

__constant__ float constant_gaussian_kernel[ 25 ];

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ������� �ʴ´�
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
#define BLOCK_SIZE (1 << 4)													// CUDA Ŀ�� thread block ������
__global__ void Gaussian_kernel_no_shared(IN unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {
	const unsigned row = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned col = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned id = row * width + col;

	int r = row, c = col;
	int w0 = 5;
	int w = floor((double)w0 / 2);
	int k, l;
	double temp, mean;

	mean = 0; d_Gaussian[id] = 0;
	for (l = -w; l <= w; l++) {
		for (k = -w; k <= w; k++) {
			if ((r + l) < 0 || (r + l) >= height || (c + k) < 0 || (c + k) >= width) continue;
			else
				mean += d_bitmaps[(r + l) * width + c + k] * constant_gaussian_kernel[(l + w) * w0 + k + w];
		}
	}
	d_Gaussian[id] = mean;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
extern __shared__ unsigned char sharedBuffer[];
__global__ void Gaussian_kernel_shared(INOUT unsigned char* d_bitmaps, OUT unsigned char* d_Gaussian, long width, long height) {
	const unsigned row = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned col = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned id = row * width + col;

	int r = row, c = col;
	int w0 = 5;
	int w = floor((double)w0 / 2);
	int k, l;
	double temp, mean;

	d_Gaussian[id] = 0;
	int thread_id = (threadIdx.y + w)* (BLOCK_SIZE + 2 * w) + (threadIdx.x + w);
	sharedBuffer[thread_id] = d_bitmaps[id];
	int x;
	if (threadIdx.x < w) {
		for (l = threadIdx.y - w; l < BLOCK_SIZE + 2*w; l++) {
			for (k = threadIdx.x - w; k < 0; k++) {
				x = (l + w) * w + k + w;
				if ((r + l) < 0 || (r + l) >= height || (c + k) < 0 || (c + k) >= width) continue;
				else
					sharedBuffer[x] = d_bitmaps[(r + l) * width + c + k];
			}
		}
	}
	else if (threadIdx.x == BLOCK_SIZE - 1) {
		for (l = threadIdx.y - w; l < BLOCK_SIZE + 2 * w; l++) {
			for (k = threadIdx.x + 1; k < BLOCK_SIZE + w; k++) {
				x = (l + w) * w + k - 1;
				if ((r + l) < 0 || (r + l) >= height || (c + k) < 0 || (c + k) >= width) continue;
				else
					sharedBuffer[x] = d_bitmaps[(r + l) * width + c + k];
			}
		}
	}
	if (threadIdx.y < w) {
		for (l = threadIdx.y - w; l < 0; l++) {
			for (k = threadIdx.x - w; k < BLOCK_SIZE + 4; k++) {
				x = (l + w) * (BLOCK_SIZE + 2*w) + k + w;
				if ((r + l) < 0 || (r + l) >= height || (c + k) < 0 || (c + k) >= width) continue;
				else
					sharedBuffer[x] = d_bitmaps[(r + l) * width + c + k];
			}
		}
	}
	else if (threadIdx.y == BLOCK_SIZE - 1) {
		for (l = threadIdx.y + 1; l < BLOCK_SIZE + w; l++) {
			for (k = threadIdx.x - w; k < BLOCK_SIZE + 2*w; k++) {
				x = (l - 1) * (BLOCK_SIZE + 2*w) + k + w;
				if ((r + l) < 0 || (r + l) >= height || (c + k) < 0 || (c + k) >= width) continue;
				else
					sharedBuffer[x] = d_bitmaps[(r + l) * width + c + k];
			}
		}
	}
	__syncthreads();
	mean = 0;
	for (k = -w; k <= w; k++) {
		for (l = -w; l <= w; l++) {
			mean += sharedBuffer[(threadIdx.y + 2 + k) * (BLOCK_SIZE + 4) + (threadIdx.x + 2 + l)] * constant_gaussian_kernel[(k + w) * w0 + w + l];
		}
	}

	d_Gaussian[id] = mean;

}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Constant variable �� gaussian kernel�� �����ϴ� �Լ�
//	�Ŀ� gaussian filtering ���� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Set_Gaussain_Kernel(){
	float _1 = 1.0f / 256.0f;
	float _4 = _1 * 4;
	float _6 = _1 * 6;
	float _16 = _1 * 16;
	float _24 = _1 * 24;
	float _36 = _1 * 36;

	float *p_gaussian_kernel = new float[25];

	p_gaussian_kernel[0] = p_gaussian_kernel[4] = p_gaussian_kernel[20] = p_gaussian_kernel[24] = _1;
	p_gaussian_kernel[1] = p_gaussian_kernel[3] = p_gaussian_kernel[5] = p_gaussian_kernel[9]= _4;
	p_gaussian_kernel[15] = p_gaussian_kernel[19] = p_gaussian_kernel[21] = p_gaussian_kernel[23] = _4;
	p_gaussian_kernel[2] = p_gaussian_kernel[10] = p_gaussian_kernel[14] = p_gaussian_kernel[22] = _6;
	p_gaussian_kernel[6] = p_gaussian_kernel[8] = p_gaussian_kernel[16] = p_gaussian_kernel[18] = _16;
	p_gaussian_kernel[7] = p_gaussian_kernel[11] =p_gaussian_kernel[13] = p_gaussian_kernel[17] = _24;
	p_gaussian_kernel[12] = _36;

	hipMemcpyToSymbol(HIP_SYMBOL( constant_gaussian_kernel), p_gaussian_kernel, sizeof( float ) * 25 );

	delete[] p_gaussian_kernel;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Ŀ���� �����ϱ� �� �ʿ��� �ڷ�� �غ� �� Ŀ���� ������ ����̽��� ����
//	Shared_flag �Է� �� NO_SHARED �� SHARED �� �� ���� ��ũ�θ� ������
//	flag���� �´� Ŀ���� ����
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

float Do_Gaussian_on_GPU(IN unsigned char *p_bitmaps, OUT unsigned char *p_Gaussian, long width, long height, int Shared_flag)
{
	Set_Gaussain_Kernel();
	CUDA_CALL(hipSetDevice(0));
	unsigned int total_pixel = width * height;

	unsigned char *d_bitmaps, *d_Gaussian;
	size_t mem_size;

	mem_size = width * height * sizeof(unsigned char);
	CUDA_CALL(hipMalloc(&d_bitmaps, mem_size));

	CUDA_CALL(hipMemcpy(d_bitmaps, p_bitmaps, mem_size, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_Gaussian, mem_size));

	dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

	dim3 gridDim((width) / blockDim.x, (height) / blockDim.y); 
	CHECK_TIME_INIT_GPU();
	CHECK_TIME_START_GPU();

	switch (Shared_flag)
	{
	case NO_SHARED:
		Gaussian_kernel_no_shared << <gridDim, blockDim >> > (d_bitmaps, d_Gaussian, width, height);
		break;
	case SHARED:
		Gaussian_kernel_shared << < gridDim, blockDim, sizeof(unsigned char) * (BLOCK_SIZE + 2 * Window) * (BLOCK_SIZE + 2 * Window)>> > (d_bitmaps, d_Gaussian, width, height);
		break;
	}

	CUDA_CALL(hipDeviceSynchronize());
	CHECK_TIME_END_GPU(device_time);
	CHECK_TIME_DEST_GPU();

	CUDA_CALL(hipMemcpy(p_Gaussian, d_Gaussian, mem_size, hipMemcpyDeviceToHost));
	
	hipFree(d_bitmaps);
	hipFree(d_Gaussian);

	return device_time;
}