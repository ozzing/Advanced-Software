#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
    CUDA_CALL(hipEventCreate(&cuda_timer_start));
    CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
    CUDA_CALL(hipEventDestroy(cuda_timer_start));
    CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
    hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
    TIMER_T ms;
    hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
    hipEventSynchronize(cuda_timer_stop);

    hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
    return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define BLOCK_SIZE 32
#define N_EQUATIONS (1 << 13)

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int N;
float* A, * B, * C;
float* X0, * X1, * FX0, * FX1;
float* X0_GPU, * X1_GPU, * FX0_GPU, * FX1_GPU;

hipError_t find_roots_GPU(float* A, float* B, float* C,
    float* X0, float* X1, float* FX0, float* FX1, int n);

__global__ void find_roots_Kernel(float* A, float* B, float* C,
    float* X0, float* X1, float* FX0, float* FX1, int n)
{
    /*Thread Idx*/
    int row;
    int col;
    int tid;
    /*Todo*/
    row = blockDim.y * blockIdx.y + threadIdx.y;
    tid = col = blockDim.x * blockIdx.x + threadIdx.x;
    float a, b, c, d, x0, x1, tmp;

    a = A[tid]; b = B[tid]; c = C[tid];
    d = sqrtf(b * b - 4.0f * a * c);
    tmp = 1.0f / (2.0f * a);
    x0 = (-b - d) * tmp;
    x1 = (-b + d) * tmp;
    if (x0 > x1) {
        int temp = x1;
        x1 = x0;
        x0 = temp;
    }
    X0[tid] = x0; X1[tid] = x1;
    FX0[tid] = (a * x0 + b) * x0 + c;
    FX1[tid] = (a * x1 + b) * x1 + c;

}


void find_roots_CPU(float* A, float* B, float* C,
    float* X0, float* X1, float* FX0, float* FX1, int n) {
    int i;
    float a, b, c, d, x0, x1, tmp;
    for (i = 0; i < n; i++) {
        a = A[i]; b = B[i]; c = C[i];
        d = sqrtf(b * b - 4.0f * a * c);
        tmp = 1.0f / (2.0f * a);
        x0 = (-b - d) * tmp;
        x1 = (-b + d) * tmp;
        if (x0 > x1) {
            int temp = x1;
            x1 = x0;
            x0 = temp;
        }
        X0[i] = x0; X1[i] = x1;
        FX0[i] = (a * x0 + b) * x0 + c;
        FX1[i] = (a * x1 + b) * x1 + c;
    }
}

void init_bin_file(int size) {
    srand((unsigned)time(NULL));
    printf("***Binary File init Start!!\n");
   
    FILE* fp = fopen("A.bin", "wb");
    fwrite(&size, sizeof(int), 1, fp);
    float random_num;
    for (int i = 0; i < size; i++) {
        random_num = (float)(((double)rand() / RAND_MAX) * 40.0f);
        fwrite(&random_num, sizeof(float), 1, fp);
    }
    fclose(fp);
   
    fp = fopen("B.bin", "wb");
    fwrite(&size, sizeof(int), 1, fp);
    for (int i = 0; i < size; i++) {
        random_num = (float)(((double)rand() / RAND_MAX) * 40.0f);
        fwrite(&random_num, sizeof(float), 1, fp);
    }
    fclose(fp);
    
    fp = fopen("C.bin", "wb");
    fwrite(&size, sizeof(int), 1, fp);
    for (int i = 0; i < size; i++) {
        random_num = (float)(((double)rand() / RAND_MAX) * 40.0f);
        fwrite(&random_num, sizeof(float), 1, fp);
    }
    fclose(fp);

    printf("***Binary File init End!!\n\n");
}

void read_bin_file()
{
    printf("***Binary File Read Start!!\n");
    /*Todo*/
    A = new float[N_EQUATIONS];
    B = new float[N_EQUATIONS];
    C = new float[N_EQUATIONS];
    X0 = new float[N_EQUATIONS];
    X1 = new float[N_EQUATIONS];
    FX0 = new float[N_EQUATIONS];
    FX1 = new float[N_EQUATIONS];
    X0_GPU = new float[N_EQUATIONS];
    X1_GPU = new float[N_EQUATIONS];
    FX0_GPU = new float[N_EQUATIONS];
    FX1_GPU = new float[N_EQUATIONS];

    FILE* fp = fopen("A.bin", "rb");
    fread(&N, sizeof(int), 1, fp);
    fread(A, sizeof(float), N_EQUATIONS, fp);
    fclose(fp);

    fp = fopen("B.bin", "rb");
    fread(&N, sizeof(int), 1, fp);
    fread(B, sizeof(float), N_EQUATIONS, fp);
    fclose(fp);

    fp = fopen("C.bin", "rb");
    fread(&N, sizeof(int), 1, fp);
    fread(C, sizeof(float), N_EQUATIONS, fp);
    fclose(fp);

    printf("***Binary File Read End!!\n\n");
}

int main()
{
    init_bin_file(N_EQUATIONS);
    read_bin_file();

    printf("The problem size is %d.\n", N);
    int i;
    FILE* fp;

    //CPU 
    printf("***Fibonacci_CPU Start!!\n");
    CHECK_TIME_START;
    find_roots_CPU(A, B, C, X0, X1, FX0, FX1, N);
    CHECK_TIME_END(compute_time);
    printf("***Fibonacci_CPU End!!\n\n");

    printf("CPU time = %.6f\n", compute_time);
    //GPU

    printf("***Fibonacci_GPU Start!!\n");
    find_roots_GPU(A, B, C, X0, X1, FX0_GPU, FX1_GPU, N);
    printf("***Fibonacci_GPU End!!\n\n");

    printf("GPU time = %.6f\n", device_time);

    for (i = 0; i < N; i++)
    {
        if (FX0[i] != FX0_GPU[i] && !isnan(FX0[i]) && !isnan(FX0_GPU[i]))
        {
            printf("%f != %f\n", FX0[i], FX0_GPU[i]);
            break;
        }
        if (FX1[i] != FX1_GPU[i] && !isnan(FX1[i]) && !isnan(FX1_GPU[i]))
        {
            printf("%f != %f\n", FX1[i], FX1_GPU[i]);
            break;
        }
    }

    if (i == N)
        printf("***Kernel execution Success!!\n\n");

    //  Write the output array into the output file.
    fp = fopen("X0.bin", "wb");
    if (!fp) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }
    fwrite(&N, sizeof(int), 1, fp);
    for (i = 0; i < N; i++)
        fwrite(&X0_GPU[i], sizeof(float), 1, fp);
    fclose(fp);

    fp = fopen("X1.bin", "wb");
    if (!fp) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }
    fwrite(&N, sizeof(int), 1, fp);
    for (i = 0; i < N; i++)
        fwrite(&X1_GPU[i], sizeof(float), 1, fp);
    fclose(fp);

    fp = fopen("FX0.bin", "wb");
    if (!fp) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }
    fwrite(&N, sizeof(int), 1, fp);
    for (i = 0; i < N; i++)
        fwrite(&FX0_GPU[i], sizeof(float), 1, fp);
    fclose(fp);


    fp = fopen("FX1.bin", "wb");
    if (!fp) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }
    fwrite(&N, sizeof(int), 1, fp);
    for (i = 0; i < N; i++)
        fwrite(&FX1_GPU[i], sizeof(float), 1, fp);
    fclose(fp);

    printf("end!!\n\n");
    return 0;
}


hipError_t find_roots_GPU(float* A, float* B, float* C,
    float* X0, float* X1, float* FX0, float* FX1, int n)
{
    CHECK_TIME_INIT_GPU();

    hipError_t cudaStatus;
    /*Todo*/
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    hipDeviceProp_t deviceProp;
    CUDA_CALL(hipGetDeviceProperties(&deviceProp, 0));

    float* d_A, * d_B, * d_C;
    float* d_X0, * d_X1, * d_FX0, * d_FX1;
    float* d_X0_GPU, * d_X1_GPU, * d_FX0_GPU, * d_FX1_GPU;
    size_t size = sizeof(float) * N;

    CUDA_CALL(hipMalloc(&d_A, size))
    CUDA_CALL(hipMalloc(&d_B, size))
    CUDA_CALL(hipMalloc(&d_C, size))
    CUDA_CALL(hipMalloc(&d_X0, size))
    CUDA_CALL(hipMalloc(&d_X1, size))
    CUDA_CALL(hipMalloc(&d_FX0, size))
    CUDA_CALL(hipMalloc(&d_FX1, size))
    
    CUDA_CALL(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice))
    CUDA_CALL(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice))
    CUDA_CALL(hipMemcpy(d_C, C, size, hipMemcpyHostToDevice))


    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(N_EQUATIONS / dimBlock.x);
    CHECK_TIME_START_GPU();
    find_roots_Kernel <<< dimGrid, dimBlock >>> (d_A, d_B, d_C, d_X0, d_X1, d_FX0, d_FX1, N);
    CHECK_TIME_END_GPU(device_time);

    CUDA_CALL(hipDeviceSynchronize())
    CUDA_CALL(hipMemcpy(X0_GPU, d_X0, size, hipMemcpyDeviceToHost))
    CUDA_CALL(hipMemcpy(X1_GPU, d_X1, size, hipMemcpyDeviceToHost))
    CUDA_CALL(hipMemcpy(FX0_GPU, d_FX0, size, hipMemcpyDeviceToHost))
    CUDA_CALL(hipMemcpy(FX1_GPU, d_FX1, size, hipMemcpyDeviceToHost))


        Error:
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_X0);
    hipFree(d_X1);
    hipFree(d_FX0);
    hipFree(d_FX1);

    CHECK_TIME_DEST_GPU();
    return cudaStatus;
}